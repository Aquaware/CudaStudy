#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>

#include <hip/hip_runtime.h>

#include "kernels.h"



using namespace std;

__global__
void addX(float* a, float* b, float* c)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	c[i] = a[i] + b[i];
}



void cu() {
	int size = 100;
	float* a = new float[size];
	float* b = new float[size];
	float* c = new float[size];

	for (int i = 0; i < size; i++) {
		a[i] = 1.0f;
		b[i] = 200.0f;
		c[i] = 0.0f;
	}

	int bytes = size * sizeof(float);

	float* dA;
	float* dB;
	float* dC;

	hipError_t ret;
	ret = hipMalloc(&dA, bytes);
	cout << "ret: " << ret << endl;
	ret = hipMalloc(&dB, bytes);
	cout << "ret: " << ret << endl;
	ret = hipMalloc(&dC, bytes);
	cout << "ret: " << ret << endl;

	ret = hipMemcpy(dA, a, bytes, hipMemcpyHostToDevice);
	cout << "ret: " << ret << endl;

	ret = hipMemcpy(dB, b, bytes, hipMemcpyHostToDevice);
	cout << "ret: " << ret << endl;

	ret = hipMemcpy(dC, c, bytes, hipMemcpyHostToDevice);
	cout << "ret: " << ret << endl;

	dim3 block = dim3(16);
	dim3 grid = dim3((size + block.x - 1) / block.x);
	addX<<<grid, block>>>(dA, dB, dC);

	ret = hipDeviceSynchronize();
	cout << "ret: " << ret << endl;

	ret = hipMemcpy(c, dC, bytes, hipMemcpyDeviceToHost);
	cout << "ret: " << ret << endl;

	for (int i = 0; i < 5; i++) {
		cout << "a: " << a[i] << " b: " << b[i] << " c: " << c[i] << endl;
	}

	ret = hipFree(dA);
	cout << "ret: " << ret << endl;

	ret = hipFree(dB);
	cout << "ret: " << ret << endl;

	ret = hipFree(dC);
	cout << "ret: " << ret << endl;

	delete[] a;
	delete[] b;
	delete[] c;
}
